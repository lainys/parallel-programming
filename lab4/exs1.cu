#include "hip/hip_runtime.h"
# include <time.h>
# include <stdlib.h>
# include <stdio.h>
# include <string.h>
# include <hip/hip_runtime.h>
# include <ctime>
#include <hip/hip_runtime.h>
#include "./common/inc/helper_image.h"

float checkGPU(unsigned char * d_result_pixels, int radius, int k);
float cudaPallel(unsigned char * d_result_pixels, int radius);

texture<unsigned char, 2, hipReadModeElementType> g_Texture;
unsigned int width = 512, height = 512;


__global__  void BoxBlur_kernel(unsigned char * pDst, int radius, int w, int h)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y;
	// ��������, ��� ������� ������� �� ������� �� ������� ����������� 
	if (tidx < w && tidy < h)
	{
		unsigned int r = 0;
		for (int ir = -radius; ir <= radius; ir++)
			for (int ic = -radius; ic <= radius; ic++)
			{
				r += tex2D(g_Texture, tidx + 0.5f + ic, tidy + 0.5f + ir);
			}
		//������������ ���������� ����������� 
		r /= ((2 * radius + 1)*(2 * radius + 1));
		pDst[tidx + tidy * w] = (unsigned char)r;
	}
}

void loadImage(char *file, unsigned char** pixels, unsigned int * width, unsigned int * height)
{
	size_t file_length = strlen(file);

	if (!strcmp(&file[file_length - 3], "pgm"))
	{
		if (sdkLoadPGM<unsigned char>(file, pixels, width, height) != true)
		{
			printf("Failed to load PGM image file: %s\n", file);
			exit(EXIT_FAILURE);
		}
	}
	return;
}

void saveImage(char *file, unsigned char* pixels, unsigned int width, unsigned int  height)
{
	size_t file_length = strlen(file);
	if (!strcmp(&file[file_length - 3], "pgm"))
	{
		sdkSavePGM(file, pixels, width, height);
	}
	return;
}

int main(int argc, char ** argv)
{
	unsigned char * d_result_pixels;
	unsigned char * h_result_pixels;
	unsigned char * h_pixels = NULL;
	unsigned char * d_pixels = NULL;

	int radius = 8;

	char * src_path = "lena.pgm";
	char * d_result_path = "lena_box_blur.pgm";

	loadImage(src_path, &h_pixels, &width, &height);

	printf("Image size %dx%d\n", width, height);

	int image_size = sizeof(unsigned char) * width * height;

	h_result_pixels = (unsigned char *)malloc(image_size);
	hipMalloc((void **)& d_pixels, image_size);
	hipMalloc((void **)& d_result_pixels, image_size);
	hipMemcpy(d_pixels, h_pixels, image_size, hipMemcpyHostToDevice);


	hipChannelFormatDesc desc = hipCreateChannelDesc<uchar1>();
	hipError_t error = hipBindTexture2D(0, &g_Texture, d_pixels, &desc, width, height, width * sizeof(unsigned char));

	if (hipSuccess != error) {
		printf("ERROR: Failed to bind texture.\n");
		exit(-1);
	}
	else {
		printf("Texture was successfully binded\n");
	}

	int N = radius - 1;

	int* rs = new int[N];
	float* ans1 = new float[N];

	for (int i = 1; i < radius; i++) {
		rs[i-1] = i;
		ans1[i-1] = checkGPU(d_result_pixels,i,100);
		std::cout << i << std::endl;
	}

	std::ofstream out("text1.txt", 'w');

	for (int i = 0; i < N; i++) {
		out << rs[i] << " ";
	}
	out << std::endl;


	for (int i = 0; i < N; i++) {
		out << ans1[i] << " ";
	}
	out << std::endl;


	out.close();
	

	hipMemcpy(h_result_pixels, d_result_pixels, image_size, hipMemcpyDeviceToHost);
	saveImage(d_result_path, h_result_pixels, width, height);
	hipUnbindTexture(&g_Texture);

	hipFree(d_pixels);
	hipFree(d_result_pixels);

	delete rs, ans1;

	return 0;
}

float checkGPU(unsigned char * d_result_pixels,int radius, int k) {
	float time = 0;

	for (int i = 0; i < k; i++) {
		time += cudaPallel(d_result_pixels,radius);
	}

	return time / (1000.0f * k);
}

float cudaPallel(unsigned char * d_result_pixels,int radius) {

	int n = 16;
	dim3 block(n, n);
	dim3 grid(width / n, height / n);
	
	//----
	hipEvent_t start, stop;
	float gpuTime = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
	//----
	BoxBlur_kernel << < grid, block >> >(d_result_pixels, radius, width, height);
	//negative_kernel << < grid, block >> >(d_result_pixels, width, height);

	//----
	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);

	//printf("N, time spent executing by the GPU: %.5f ms\n", gpuTime);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	//----
	/* CUDA method */

	return gpuTime;
}
