#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#define	N	(1024*1024)

__global__ void kernel(float * data)
{
	int   idx = blockIdx.x * blockDim.x + threadIdx.x;
	float x = 2.0f * 3.1415926f * (float)idx / (float)N;
	data[idx] = sinf(sqrtf(x));

}

int main(int argc, char *argv[])
{
	int		deviceCount;
	hipDeviceProp_t	devProp;
	int blocks[3];
	int grids[3];

	hipGetDeviceCount(&deviceCount);

	for (int device = 0; device < deviceCount; device++)
	{
		hipGetDeviceProperties(&devProp, device);
		blocks[0] = devProp.maxThreadsDim[0];
		blocks[1] = devProp.maxThreadsDim[1];
		blocks[2] = devProp.maxThreadsDim[2];

		grids[0] = devProp.maxGridSize[0];
		grids[1] = devProp.maxGridSize[1];
		grids[2] = devProp.maxGridSize[2];
	}


	float *  a = (float*)malloc(N * sizeof(float));
	float * dev = nullptr;
	// �������� ������ �� GPU
	hipMalloc((void**)&dev, N * sizeof(float));

	hipEvent_t start, stop;		//��������� ���������� ����  hipEvent_t 
	float       gpuTime = 0.0f;
	// ������� ������� ������ � ��������� ���������� ���� 
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//����������� ������� start  � ������� ����� 
	hipEventRecord(start, 0);
	// ������� ���� 
	kernel <<<dim3(grids[0],grids[1],grids[2]), dim3(blocks[0],blocks[1],blocks[2]) >> > (dev);
	//����������� ������� stop  � ������� ����� 
	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	// ����������� ����� ����� ��������� 
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("time spent executing by the GPU: %.5f ms\n", gpuTime);
	// ���������� ��������� ������� 
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(a, dev, N * sizeof(float), hipMemcpyDeviceToHost);
	// ���������� ���������� ������
	hipFree(dev);

	free(a);


	return 0;
}
