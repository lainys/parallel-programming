#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>

__global__ void kernel(int* a, int* b, int* c)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	c[idx] = a[idx] + b[idx];
	
}

template<typename T>
T init(T a,int N) {

	for (int i = 0; i < N; i++) {
		a[i] = 1;
	}
	return a;
}

int main(int argc, char *argv[])
{
	int N = 10;

	int* a = (int*) malloc(N * sizeof(int));
	int* b = (int*) malloc(N * sizeof(int));
	int* c = (int*) malloc(N * sizeof(int));

	int* dA = nullptr;
	int* dB = nullptr;
	int* dC = nullptr;

	a = init(a, N);
	b = init(b, N);

	hipMalloc((void**)&dA, N * sizeof(int));
	hipMalloc((void**)&dB, N * sizeof(int));
	hipMalloc((void**)&dC, N * sizeof(int));

	hipMemcpy(dA, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dB, b, N * sizeof(int), hipMemcpyHostToDevice);
	
	hipEvent_t start, stop;		
	float gpuTime = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	kernel <<<dim3(N/10,1), dim3(10,1)>>> (dA,dB,dC);

	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);

	hipEventElapsedTime(&gpuTime, start, stop);
	printf("time spent executing by the GPU: %.5f ms\n", gpuTime);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(c, dC, N * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);


	for (int i = 0; i < N; i++) {
		printf("%d\n", c[i]);
	}

	free(a);
	free(b);
	free(c);

	return 0;
}
