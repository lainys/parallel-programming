#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#define	N	(1024*1024)

__global__ void kernel(float * data)
{
	int   idx = blockIdx.x * blockDim.x + threadIdx.x;
	float x = 2.0f * 3.1415926f * (float)idx / (float)N;
	data[idx] = sinf(sqrtf(x));
}

int main(int argc, char *argv[])
{
	float *  a = (float*)malloc(N * sizeof(float));
	float * dev = nullptr;
	// �������� ������ �� GPU
	hipMalloc((void**)&dev, N * sizeof(float));

	hipEvent_t start, stop;		//��������� ���������� ����  hipEvent_t 
	float       gpuTime = 0.0f;
	// ������� ������� ������ � ��������� ���������� ���� 
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//����������� ������� start  � ������� ����� 
	hipEventRecord(start, 0);
	// ������� ���� 
	kernel << <dim3((N / 512), 1), dim3(512, 1) >> > (dev);
	//����������� ������� stop  � ������� ����� 
	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	// ����������� ����� ����� ��������� 
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("time spent executing by the GPU: %.5f ms\n", gpuTime);
	// ���������� ��������� ������� 
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(a, dev, N * sizeof(float), hipMemcpyDeviceToHost);
	// ���������� ���������� ������
	hipFree(dev);

	free(a);


	return 0;
}
